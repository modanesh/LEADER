#include "hip/hip_runtime.h"
#include <despot/GPUinterface/GPUlower_bound.h>
#include <despot/GPUcore/GPUbuiltin_lower_bound.h>

#include <despot/GPUinterface/GPUpomdp.h>

using namespace std;

namespace despot {

/* =============================================================================
 * TrivialParticleLowerBound class
 * =============================================================================*/


DEVICE Dvc_ValuedAction Dvc_TrivialParticleLowerBound::Value(
		int scenarioID, Dvc_State * particles) {
	Dvc_ValuedAction va = DvcModelGetBestAction_();
	va.value *= 1.0 / (1 - Dvc_Globals::Dvc_Discount(Dvc_config));
	return va;
}

__global__ void PassLbValueFunc(Dvc_TrivialParticleLowerBound* lowerbound)
{
	DvcParticleLowerBound_Value_=&(lowerbound->Value);
}

} // namespace despot
